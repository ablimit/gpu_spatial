#include "hip/hip_runtime.h"

#include "postgres.h"
#include "lwgeom_functions_analytic.h"

__global__ void contains(PG_FUNCTION_ARGS)
{

  GSERALIZED *geom1, *geom2;
  GEOSGeometry *g, *h;
  GBOX b1, b2;
  int type1, type2;
  LWGEOM *lwg;
  LWPOINT *lwp;
  bool result;

  geom1= (GSERIALIZED *) PG_DETOAST_DATUM(PG_GETARG_DATUM(0));
  geom2= (GSERIALIZED *) PG_DETOAST_DATUM(PG_GETARG_DATUM(1));

  lwp = lwgeom_as_point(lwgeom_from_gseralized(geom2));

  result = point_in_polygon((LWPOLY*) lwg, lwp);

  PG_RETURN_BOOL(result);

}
