//
//
//	gpu-info
//
//		Returns information on the installed GPU device.
//
//
//	

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cstring>
#include <cstdio>


#define  VER	"0.1"


char* device_info(void)
{
	hipDeviceProp_t		prop;
	int					count;
	hipError_t			result;
	char				*buffer;
	
	buffer = (char*)malloc(1024);
	if( buffer ) {
	
		result = hipGetDeviceCount(&count);
		if( result == hipSuccess ) {

			if( count > 0 ) {
				result = hipGetDeviceProperties(&prop, 0);
				if( result == hipSuccess )
					sprintf(buffer, "%s (%d.%d), count: %d, lib ver: %s", 
							prop.name, prop.major, prop.minor, count, VER);
			} else {
				sprintf(buffer, "No gpu devices detected");
			}
		}
	}
	return buffer;
}





//
//
//	 C wrapper for the Postgres plugin
//
//
extern "C" {

char* gi_device_info(void) 
{
	return device_info();
}

}
